// (c) Meta Platforms, Inc. and affiliates. Confidential and proprietary.

#include <stdio.h>

#include "kineto_playground.cuh"


namespace kineto {

void warmup(void) {
  // Inititalizing CUDA can take a while which we normally do not want to see in Kineto traces.
  // This is done in various ways that take Kineto as dependency. This is our way of doing warmup
  // for kineto_playground
	size_t bytes = 1000;
	float* mem = NULL;
	auto error = hipMalloc(&mem, bytes);
  if (error != hipSuccess) {
    printf("hipMalloc failed during kineto_playground warmup. error code: %d", error);
    return;
  }

  hipFree(mem); 
}

void basicMemcpyMemset(void) {
  size_t size = (1 << 8) * sizeof(float);
  float *hostMemSrc, *deviceMem, *hostMemDst;
  hipError_t err;

  hostMemSrc = (float*)malloc(size);
  hostMemDst = (float*)malloc(size);
  err = hipMalloc(&deviceMem, size);
  if (err != hipSuccess) {
    printf("hipMalloc failed during %s", __func__);
    return;
  }

  memset(hostMemSrc, 1, size);
  hipMemcpy(deviceMem, hostMemSrc, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("hipMemcpy failed during %s", __func__);
    return;
  }

  hipMemcpy(hostMemDst, deviceMem, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("hipMemcpy failed during %s", __func__);
    return;
  }

  free(hostMemSrc);
  free(hostMemDst);
  hipFree(deviceMem);
}

void playground(void) {
  // Add your experimental CUDA implementation here. 
}

}
