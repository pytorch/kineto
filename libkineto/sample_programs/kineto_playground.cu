#include "hip/hip_runtime.h"
/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <stdio.h>

#include "kineto_playground.cuh"


namespace kineto {

void warmup(void) {
  // Inititalizing CUDA can take a while which we normally do not want to see in Kineto traces.
  // This is done in various ways that take Kineto as dependency. This is our way of doing warmup
  // for kineto_playground
	size_t bytes = 1000;
	float* mem = NULL;
	auto error = hipMalloc(&mem, bytes);
  if (error != hipSuccess) {
    printf("hipMalloc failed during kineto_playground warmup. error code: %d", error);
    return;
  }

  hipFree(mem);
}

float *hA, *dA, *hOut;
int num = 50'000;

void basicMemcpyToDevice(void) {
  size_t size = num * sizeof(float);
  hipError_t err;

  hA = (float*)malloc(size);
  hOut = (float*)malloc(size);
  err = hipMalloc(&dA, size);
  if (err != hipSuccess) {
    printf("hipMalloc failed during %s", __func__);
    return;
  }

  memset(hA, 1, size);
  err = hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("hipMemcpy failed during %s", __func__);
    return;
  }
}

void basicMemcpyFromDevice(void) {

  size_t size = num * sizeof(float);
  hipError_t err;

  err = hipMemcpy(hOut, dA, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("hipMemcpy failed during %s", __func__);
    return;
  }

  free(hA);
  free(hOut);
  hipFree(dA);
}

__global__ void square(float* A, int N) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    A[i] *= A[i];
  }
}

void playground(void) {
  // Add your experimental CUDA implementation here.
}

void compute(void) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
  for (int i = 0; i < 10; i++) {
    square<<<blocksPerGrid, threadsPerBlock>>> (dA, num);
  }
}

} // namespace kineto
